#include "hip/hip_runtime.h"
#include "imgproc_legacy.h"

template <typename Ptr2D>
__global__ void copyMakeBroderKernel(const Ptr2D src, Ptr2D dst, int top, int bottom, int left, int right,
                                     int borderType, const int value)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int x_shift = x - left;
    const int y_shift = y - top;
    int out_height = dst.rows;
    int out_width = dst.cols;
    if (x < out_width && y < out_height)
    {
        dst[y][x] = 0;
    }
}

template <typename T, int cn> // uchar3 float3 uchar float
void copyMakeBorderImpl(const T src, T dst, int top, int bottom, int left, int right, int borderType, const int value)
{
    // typedef void (*func_t)(const Ptr2D src, Ptr2D dst, int top, int bottom, int left, int right, int borderType)
}

namespace cucv::legacy::cuda
{



} // namespace cucv::legacy::cuda
